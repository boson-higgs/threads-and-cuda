#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "pic_type.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale( CudaPic t_color_pic, CudaPic t_bw_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	// Get point from color picture
	uchar3 l_bgr = t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ];

	// Store BW point to new image
	t_bw_pic.m_p_uchar1[ l_y * t_bw_pic.m_size.x + l_x ].x = l_bgr.x * 0.11 + l_bgr.y * 0.59 + l_bgr.z * 0.30;
}

void cu_run_grayscale( CudaPic t_color_pic, CudaPic t_bw_pic )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 16;
	dim3 l_blocks( ( t_color_pic.m_size.x + l_block_size - 1 ) / l_block_size, ( t_color_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_grayscale<<< l_blocks, l_threads >>>( t_color_pic, t_bw_pic );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}
