#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "pic_type.h"
#include "animation.h"

// Demo kernel to create chess board
__global__ void kernel_creategradient( CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	int l_dy = l_x * t_color_pic.m_size.y / t_color_pic.m_size.x + l_y - t_color_pic.m_size.y;
	unsigned char l_color = 255 * abs( l_dy ) / t_color_pic.m_size.y;

	uchar3 l_bgr = ( l_dy < 0 ) ? ( uchar3 ) { l_color, 255 - l_color, 0 } : ( uchar3 ) { 0, 255 - l_color, l_color };

	// Store point into image
	t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ] = l_bgr;
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_pic.m_size.y ) return;
	if ( l_x >= t_small_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_pic.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_pic.m_size.x || l_bx < 0 ) return;

	// Get point from small image
	uchar4 l_fg_bgra = t_small_pic.m_p_uchar4[ l_y * t_small_pic.m_size.x + l_x ];
	uchar3 l_bg_bgr = t_big_pic.m_p_uchar3[ l_by * t_big_pic.m_size.x + l_bx ];
	uchar3 l_bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

	// Store point into image
	t_big_pic.m_p_uchar3[ l_by * t_big_pic.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( t_big_pic, t_small_pic, t_position );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

void Animation::start( CudaPic t_bg_pic, CudaPic t_ins_pic )
{
	if ( m_initialized ) return;
	hipError_t l_cerr;

	m_cuda_bg_pic = t_bg_pic;
	m_cuda_res_pic = t_bg_pic;
	m_cuda_ins_pic = t_ins_pic;

	// Memory allocation in GPU device
	// Memory for background
	l_cerr = hipMalloc( &m_cuda_bg_pic.m_p_void, m_cuda_bg_pic.m_size.x * m_cuda_bg_pic.m_size.y * sizeof( uchar3 ) );
	if ( l_cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	// Creation of background gradient
	int l_block_size = 32;
	dim3 l_blocks( ( m_cuda_bg_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_cuda_bg_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_creategradient<<< l_blocks, l_threads >>>( m_cuda_bg_pic );

	m_initialized = 1;
}

void Animation::next( CudaPic t_res_pic, int2 t_position )
{
	if ( !m_initialized ) return;

	hipError_t cerr;

	// Copy data internally GPU from background into result
	cerr = hipMemcpy( m_cuda_res_pic.m_p_void, m_cuda_bg_pic.m_p_void, m_cuda_bg_pic.m_size.x * m_cuda_bg_pic.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// insert picture
	int l_block_size = 32;
	dim3 l_blocks( ( m_cuda_ins_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_cuda_ins_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( m_cuda_res_pic, m_cuda_ins_pic, t_position );

	// Copy data to GPU device
	cerr = hipMemcpy( t_res_pic.m_p_void, m_cuda_res_pic.m_p_void, m_cuda_res_pic.m_size.x * m_cuda_res_pic.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

}

void Animation::stop()
{
	if ( !m_initialized ) return;

	hipFree( m_cuda_bg_pic.m_p_void );
	hipFree( m_cuda_res_pic.m_p_void );
	hipFree( m_cuda_ins_pic.m_p_void );

	m_initialized = 0;
}


