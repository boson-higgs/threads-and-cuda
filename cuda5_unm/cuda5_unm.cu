#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "pic_type.h"

// Demo kernel to create chess board
__global__ void kernel_chessboard( CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	unsigned char b_or_w = 255 * ( ( blockIdx.x + blockIdx.y ) & 1 );

	// Store point into image
	t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ] = { b_or_w, b_or_w, b_or_w };
}

void cu_create_chessboard( CudaPic t_color_pic, int t_square_size )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	dim3 l_blocks( ( t_color_pic.m_size.x + t_square_size - 1 ) / t_square_size,
			       ( t_color_pic.m_size.y + t_square_size - 1 ) / t_square_size );
	dim3 l_threads( t_square_size, t_square_size );
	kernel_chessboard<<< l_blocks, l_threads >>>( t_color_pic );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_alphaimg( CudaPic t_color_pic, uchar3 t_color )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	int l_diagonal = sqrtf( t_color_pic.m_size.x * t_color_pic.m_size.x + t_color_pic.m_size.y * t_color_pic.m_size.y );
	int l_dx = l_x - t_color_pic.m_size.x / 2;
	int l_dy = l_y - t_color_pic.m_size.y / 2;
	int l_dxy = sqrtf( l_dx * l_dx + l_dy * l_dy ) - l_diagonal / 2;

	// Store point into image
	t_color_pic.m_p_uchar4[ l_y * t_color_pic.m_size.x + l_x ] =
		{ t_color.x, t_color.y, t_color.z, ( unsigned char ) ( 255 - 255 * l_dxy / ( l_diagonal / 2 ) ) };
}

void cu_create_alphaimg( CudaPic t_color_pic, uchar3 t_color )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_color_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_color_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_alphaimg<<< l_blocks, l_threads >>>( t_color_pic, t_color );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_pic.m_size.y ) return;
	if ( l_x >= t_small_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_pic.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_pic.m_size.x || l_bx < 0 ) return;

	// Get point from small image
	uchar4 l_fg_bgra = t_small_pic.m_p_uchar4[ l_y * t_small_pic.m_size.x + l_x ];
	uchar3 l_bg_bgr = t_big_pic.m_p_uchar3[ l_by * t_big_pic.m_size.x + l_bx ];
	uchar3 l_bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

	// Store point into image
	t_big_pic.m_p_uchar3[ l_by * t_big_pic.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( t_big_pic, t_small_pic, t_position );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}
